#include <hip/hip_runtime.h>
   // for >2010 / cuda>5.5
#include<iostream>

using namespace std;

__global__ void addInt(int *a,int *b){
        a[0] += b[0];
        }


int main()
{
    int a =5, b=9;
    int *pa, *pb;   //device pointers
    hipMalloc(&pa,sizeof(int));  // for actual thing wrap with if != cudaSuccess
    hipMalloc(&pb,sizeof(int));
    
    hipMemcpy(pa, &a, sizeof(int), hipMemcpyHostToDevice);  // dest, point to source, size, type
    hipMemcpy(pb, &b, sizeof(int), hipMemcpyHostToDevice);

    addInt<<<1,1>>>(pa, pb);    // <<<blocks, threadsPerBlock>>>
    hipDeviceSynchronize();  //wait for gpu
    hipMemcpy(&a ,pa ,sizeof(int) , hipMemcpyDeviceToHost);

    cout<<a;
    hipFree(pa);
    hipFree(pb);
    return 0;
}