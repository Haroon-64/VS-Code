#include <iostream>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
  for(int i = 0; i < n; i++) {
    out[i] = a[i] + b[i];
  }
}

int main() {
  float *a, *b, *out;
  float *d_a, *d_b, *d_out;

  // Allocate host memory
  a   = new float[N];
  b   = new float[N];
  out = new float[N];

  // Initialize host arrays
  for(int i = 0; i < N; i++) {
    a[i] = 1.0f;
    b[i] = 2.0f;
  }

  // Allocate device memory
  hipMalloc((void**)&d_a, sizeof(float) * N);
  hipMalloc((void**)&d_b, sizeof(float) * N);
  hipMalloc((void**)&d_out, sizeof(float) * N);

  // Transfer data from host to device memory
  hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

  // Launch the kernel on the GPU
  vector_add<<<1,256>>>(d_out, d_a, d_b, N);

  // Transfer the result from device to host memory
  hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Print the first 10 elements of the result
  for(int i = 0; i < 10; i++) {
    std::cout << "out[" << i << "] = " << out[i] << std::endl;
  }

  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);
  delete[] a;
  delete[] b;
  delete[] out;

  return 0;
}
