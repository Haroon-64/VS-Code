#include <hip/hip_runtime.h>

//#include<list>
#include <iostream>
using namespace std;
__global__ void add(int *a, int *b, int *c) {
    int i=threadIdx.x;
    c[i]=a[i]+b[i];

}

int main() {

/*
std::list<int>::iterator* iterators = new std::list<int>::iterator[50];  //creatre array using iterator??
*/
    int x[]={1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28,29,30,31,32,33,34,35,36,37,38,39,40,41,42,43,44,45,46,47,48,49,50,51,52,53,54,55,56,57,58,59,60,61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,88,89,90,91,92,93,94,95,96,97,98,99,100};
    int y[]={1,4,9,16,25,36,49,64,81,100,121,144,169,196,225,256,289,324,361,400,441,484,529,576,625,676,729,784,841,900,961,1024,1089,1156,1225,1296,1369,1444,1521,1600,1681,1764,1849,1936,2025,2116,2209,2304,2401,2500,2601,2704,2809,2916,3025,3136,3249,3364,3481,3600,3721,3844,3969,4096,4225,4356,4489,4624,4761,4900,5041,5184,5329,5476,5625,5776,5929,6084,6241,6400,6561,6724,6889,7056,7225,7396,7569,7744,7921,8100,8281,8464,8649,8836,9025,9216,9409,9604,9801,10000};
    int z[sizeof(x)]={0};
    // create device pointers
    int *d_x=0, *d_y=0, *d_z=0;
    // allocate memory on device
    hipMalloc(&d_x, sizeof(x));
    hipMalloc(&d_y, sizeof(y));
    hipMalloc(&d_z, sizeof(z));
    // copy data from host to device
    hipMemcpy(d_x, x, sizeof(x), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(y), hipMemcpyHostToDevice);
  
  add<<<1,sizeof(x)/sizeof(int)>>>(d_x,d_y,d_z);

  hipMemcpy(z, d_z, sizeof(z), hipMemcpyDeviceToHost);
  cout<<"Result is: ";
  for(int i=0;i<sizeof(z)/sizeof(int);i++)
  {
      cout<<z[i]<<" ";
  }
    return 0;
}

